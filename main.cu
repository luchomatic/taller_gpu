#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

//
// Constantes para OpenGL
//
#define KEY_ESC 27
#define ANCHO 1920
#define ALTO 1080
#define false 0
#define true 1

//
// Constantes para Algoritmo de gravitacion
//
#define PI (3.141592653589793)
#define G 6.673e-11


// ===============
// ===== CPU =====
// ===============

//
// Estructuras y variables para Algoritmo de gravitacion
//

const int CUDA_BLK = 64; // Tamaño predeterminado de bloque de hilos CUDA

float toroide_alfa;
float toroide_theta;
float toroide_incremento;
float toroide_lado;
float toroide_r;
float toroide_R;

int delta_tiempo = 1.0f; //Intervalo de tiempo, longitud de un paso
int pasos;
int N;

//variables nuestras CPU//
float * masas;
float * cPositionX;
float * cPositionY;
float * cPositionZ;

float * cVelocityX;
float * cVelocityY;
float * cVelocityZ;

float * fuerza_totalX;
float * fuerza_totalY;
float * fuerza_totalZ;

//variables nuestras GPU//
float * gpu_masas;
float * gpu_cPositionX;
float * gpu_cPositionY;
float * gpu_cPositionZ;

float * gpu_cVelocityX;
float * gpu_cVelocityY;
float * gpu_cVelocityZ;

float * gpu_fuerza_totalX;
float * gpu_fuerza_totalY;
float * gpu_fuerza_totalZ;

double cColorR = (double) rand() / (RAND_MAX + 1.0);
double cColorG = (double) rand() / (RAND_MAX + 1.0);
double cColorB = (double) rand() / (RAND_MAX + 1.0);

//terminan las variables nuestras.//

//
// Funciones para Algoritmo de gravitacion
//

void calcularFuerzas(int N, int dt) {
    int cuerpo1, cuerpo2;
    float dif_X, dif_Y, dif_Z;
    float distancia;
    float F;

    for (cuerpo1 = 0; cuerpo1 < N - 1; cuerpo1++) {
        for (cuerpo2 = cuerpo1 + 1; cuerpo2 < N; cuerpo2++) {
            if ((cPositionX[cuerpo1] == cPositionX[cuerpo2]) && (cPositionY[cuerpo1] == cPositionY[cuerpo2]) && (cPositionZ[cuerpo1] == cPositionZ[cuerpo2]))
                continue;

            dif_X = cPositionX[cuerpo2] - cPositionX[cuerpo1];
            dif_Y = cPositionY[cuerpo2] - cPositionY[cuerpo1];
            dif_Z = cPositionZ[cuerpo2] - cPositionZ[cuerpo1];

            distancia = sqrt(dif_X * dif_X + dif_Y * dif_Y + dif_Z * dif_Z);

            F = (G * masas[cuerpo1] * masas[cuerpo2]) / (distancia * distancia);

            dif_X *= F;
            dif_Y *= F;
            dif_Z *= F;

            fuerza_totalX[cuerpo1] += dif_X;
            fuerza_totalY[cuerpo1] += dif_Y;
            fuerza_totalZ[cuerpo1] += dif_Z;

            fuerza_totalX[cuerpo2] -= dif_X;
            fuerza_totalY[cuerpo2] -= dif_Y;
            fuerza_totalZ[cuerpo2] -= dif_Z;
        }
    }
}

void moverCuerpos(int N, int dt) {
    int cuerpo;
    for (cuerpo = 0; cuerpo < N; cuerpo++) {

        fuerza_totalX[cuerpo] *= 1 / masas[cuerpo];
        fuerza_totalY[cuerpo] *= 1 / masas[cuerpo];

        cVelocityX[cuerpo] += fuerza_totalX[cuerpo] * dt;
        cVelocityY[cuerpo] += fuerza_totalY[cuerpo] * dt;

        cPositionX[cuerpo] += cVelocityX[cuerpo] * dt;
        cPositionY[cuerpo] += cVelocityY[cuerpo] * dt;

        fuerza_totalX[cuerpo] = 0.0;
        fuerza_totalY[cuerpo] = 0.0;
        fuerza_totalZ[cuerpo] = 0.0;

    }
}

void gravitacionCPU(int N, int dt) {
    calcularFuerzas(N, dt);
    moverCuerpos(N, dt);
}

void inicializarEstrella(int i, double n) {

    //todos van a tener la misma masa//
    masas[i] = 0.001 * 8;

    if ((toroide_alfa + toroide_incremento) >= 2 * M_PI) {
        toroide_alfa = 0;
        toroide_theta += toroide_incremento;
    } else {
        toroide_alfa += toroide_incremento;
    }

    cPositionX[i] = (toroide_R + toroide_r * cos(toroide_alfa)) * cos(toroide_theta);
    cPositionY[i] = (toroide_R + toroide_r * cos(toroide_alfa)) * sin(toroide_theta);
    cPositionZ[i] = toroide_r * sin(toroide_alfa);

    cVelocityX[i] = 0.0;
    cVelocityY[i] = 0.0;
    cVelocityZ[i] = 0.0;
}

void inicializarCuerpos(int N) {
    int cuerpo;
    double n = N;

    toroide_alfa = 0.0;
    toroide_theta = 0.0;
    toroide_lado = sqrt(N);
    toroide_incremento = 2 * M_PI / toroide_lado;
    toroide_r = 1.0;
    toroide_R = 2 * toroide_r;

    srand(time(NULL));

    for (cuerpo = 0; cuerpo < N; cuerpo++) {

        fuerza_totalX[cuerpo] = 0.0;
        fuerza_totalY[cuerpo] = 0.0;
        fuerza_totalZ[cuerpo] = 0.0;

        inicializarEstrella(cuerpo, n);

    }
    masas[0] = 2.0e2;
    cPositionX[0] = 0.0;
    cPositionY[0] = 0.0;
    cPositionZ[0] = 0.0;
    cVelocityX[0] = -0.000001;
    cVelocityY[0] = -0.000001;
    cVelocityZ[0] = 0.0;

    masas[1] = 1.0e1;
    cPositionX[1] = -1.0;
    cPositionY[1] = 0.0;
    cPositionZ[1] = 0.0;
    cVelocityX[1] = 0.0;
    cVelocityY[1] = 0.0001;
    cVelocityZ[1] = 0.0;

}

void finalizar(void) {
    free(masas);
    free(cPositionX);
    free(cPositionY);
    free(cPositionZ);
    free(cVelocityX);
    free(cVelocityY);
    free(cVelocityZ);
    free(fuerza_totalX);
    free(fuerza_totalY);
    free(fuerza_totalZ);
}

// ===============
// ===== GPU =====
// ===============

__global__ void kernelCalcularFuerzas(int N, int dt, float * gpu_masas, float * gpu_cPositionX, float * gpu_cPositionY, float * gpu_cPositionZ, float * gpu_cVelocityX, float * gpu_cVelocityY, float * gpu_cVelocityZ, float * gpu_fuerza_totalX, float * gpu_fuerza_totalY, float * gpu_fuerza_totalZ) {

    extern __shared__ float shared_size[];

    //averiguar bien como saltear a la siguiente posición
    float * sh_fuerza_totalX = & shared_size[0];
    float * sh_fuerza_totalY = & shared_size[1 * CUDA_BLK];
    float * sh_fuerza_totalZ = & shared_size[2 * CUDA_BLK];

    float * sh_cPositionX = & shared_size[3 * CUDA_BLK];
    float * sh_cPositionY = & shared_size[4 * CUDA_BLK];
    float * sh_cPositionZ = & shared_size[5 * CUDA_BLK];

    float * sh_cVelocityX = & shared_size[6 * CUDA_BLK];
    float * sh_cVelocityY = & shared_size[7 * CUDA_BLK];
    float * sh_cVelocityZ = & shared_size[8 * CUDA_BLK];

    float * sh_masas = & shared_size[9 * CUDA_BLK];

    //CON ESTA SOLUCIÓN, CONVIENE TENER MÁS HILOS QUE BLOQUES
    //LA MASA SE PUEDE PONER EN MEMORIA CONSTANTE Para eso habría que tenerla en tiempo de ejecución

    //DEFINO EL INDICE DE MI HILO
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    //Acceden coalescentemente a la memoria global para traer los datos a la memoria compartida
    for (i = 0; i < N; i++) {
        //OBTENGO LOS DATOS DE MASA Y POSICIÓN DE MI INDICE, INICIALIZO VECTOR DE FUERZA EN CERO

        sh_masas[threadIdx.x] = gpu_masas[idx];

        sh_cPositionX[threadIdx.x] = gpu_cPositionX[idx];
        sh_cPositionY[threadIdx.x] = gpu_cPositionY[idx];
        sh_cPositionZ[threadIdx.x] = gpu_cPositionZ[idx];

        sh_cVelocityX[threadIdx.x] = gpu_cVelocityX[idx];
        sh_cVelocityY[threadIdx.x] = gpu_cVelocityY[idx];
        sh_cVelocityZ[threadIdx.x] = gpu_cVelocityZ[idx];
    }

    __syncthreads();
    //Se sincronizan para asegurar que todos los hilos trajeron los datos a procesar

    for (i = 0; i < N; i++) {
        //Fuerzas inicializadas en cero
        sh_fuerza_totalX[i] = 0.0f;
        sh_fuerza_totalY[i] = 0.0f;
        sh_fuerza_totalZ[i] = 0.0f;

        for (int j = 0; j < N; j++) {
            float dif_X = sh_cPositionX[j] - sh_cPositionX[i];
            float dif_Y = sh_cPositionY[j] - sh_cPositionY[i];
            float dif_Z = sh_cPositionZ[j] - sh_cPositionZ[i];

            float distancia = sqrt(dif_X * dif_X + dif_Y * dif_Y + dif_Z * dif_Z);

            float F = (G * sh_masas[i] * sh_masas[j]) / (distancia * distancia);

            dif_X *= F;
            dif_Y *= F;
            dif_Z *= F;

            sh_fuerza_totalX[i] += dif_X;
            sh_fuerza_totalY[i] += dif_Y;
            sh_fuerza_totalZ[i] += dif_Z;
        }
    }

    __syncthreads();
    //Se sincronizan para asegurar que todos los hilos terminaron de procesar

    //Acceden coalescentemente a la memoria global para escribir los resultados
    for (i = 0; i < N; i++) {
        //GUARDO TODOS LOS DATOS EN LA MEMORIA COMPARTIDA, POSICIÓN, FUERZA, VELOCIDAD.
        gpu_fuerza_totalX[idx] = sh_fuerza_totalX[threadIdx.x];
        gpu_fuerza_totalY[idx] = sh_fuerza_totalY[threadIdx.x];
        gpu_fuerza_totalZ[idx] = sh_fuerza_totalZ[threadIdx.x];

        gpu_cPositionX[idx] = sh_cPositionX[threadIdx.x];
        gpu_cPositionY[idx] = sh_cPositionY[threadIdx.x];
        gpu_cPositionZ[idx] = sh_cPositionZ[threadIdx.x];

        gpu_cVelocityX[idx] = sh_cVelocityX[threadIdx.x];
        gpu_cVelocityY[idx] = sh_cVelocityY[threadIdx.x];
        gpu_cVelocityZ[idx] = sh_cVelocityZ[threadIdx.x];
    }
}

__global__ void kernelMoverCuerpos(int N, int dt, float * gpu_masas, float * gpu_cPositionX, float * gpu_cPositionY, float * gpu_cPositionZ, float * gpu_cVelocityX, float * gpu_cVelocityY, float * gpu_cVelocityZ, float * gpu_fuerza_totalX, float * gpu_fuerza_totalY, float * gpu_fuerza_totalZ) {

    //DEFINO EL INDICE DE MI HILO
    /*
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int cuerpo;
    for(cuerpo = 0; cuerpo < N; cuerpo++){

    	gpu_fuerza_totalX[cuerpo] *= 1/gpu_masas[cuerpo];
    	gpu_fuerza_totalY[cuerpo] *= 1/gpu_masas[cuerpo];


    	gpu_cVelocityX[i] += dt * gpu_fuerza_totalX[i];
        gpu_cVelocityY[i] += dt * gpu_fuerza_totalY[i];
        gpu_cVelocityZ[i] += dt * gpu_fuerza_totalZ[i];


    	gpu_cPositionX[cuerpo] += gpu_cVelocityX[cuerpo] *dt;
    	gpu_cPositionY[cuerpo] += gpu_cVelocityY[cuerpo] *dt;


    	gpu_fuerza_totalX[cuerpo] = 0.0;
    	gpu_fuerza_totalY[cuerpo] = 0.0;
    	gpu_fuerza_totalZ[cuerpo] = 0.0;

    }
    */
}

void gravitacionGPU(int N, int dt) {
    //TENEMOS QUE REESCRIBIR ESTAS DOS FUNCIONES PARA QUE ANDEN CON EL GPU

    // Bloque unidimensional de hilos (*blk_size* hilos)
    dim3 dimBlock(CUDA_BLK);

    // Grid unidimensional (*ceil(n/blk_size)* bloques)
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    kernelCalcularFuerzas << < dimGrid, dimBlock >>> (N, dt, gpu_masas, gpu_cPositionX, gpu_cPositionY, gpu_cPositionZ, gpu_cVelocityX, gpu_cVelocityY, gpu_cVelocityZ, gpu_fuerza_totalX, gpu_fuerza_totalY, gpu_fuerza_totalZ);
    hipDeviceSynchronize();

    //Al kernel de mover cuerpos, le paso todos los parametros de arreglos de la GPU?
    kernelMoverCuerpos << < dimGrid, dimBlock >>> (N, dt, gpu_masas, gpu_cPositionX, gpu_cPositionY, gpu_cPositionZ, gpu_cVelocityX, gpu_cVelocityY, gpu_cVelocityZ, gpu_fuerza_totalX, gpu_fuerza_totalY, gpu_fuerza_totalZ);
    hipDeviceSynchronize();
}

// ==================
// ===== OpenGL =====
// ==================

//
// Variables OpenGL
//
double alfa = 0.0;

// Para angulo de rotacion y direccion de la camara
float angle = 0.0;
float camAngleX = 0;
float camAngleY = 0;
float distancia = 10;
int ejes = 1;

// Vector actual que representa la direccion de la camara
float lx = 0.0f, lz = -1.0f;
// posicion XZ de la camara
float x = 0.0f, z = 5.0f;

int oldX = 0, oldY = 0;
int rotate = false;

//
// Funciones OpenGL
//

//Funcion que se llama cada vez que se quiere dibujar nuevamente en la pantalla
//Se llama cada vez que se produce el evento render
void GL_camara() {
    float camX, camY, camZ;

    //Camara mirando al origen (pickObjX,pickObjY,pickObjZ) = (0,0,0)
    float pickObjX = 0.0;
    float pickObjY = 0.0;
    float pickObjZ = 0.0;

    camX = distancia * sin(camAngleX);
    camY = distancia * sin(camAngleY);
    camZ = distancia * cos(camAngleY) * cos(camAngleX);

    //Ubicar la camara
    gluLookAt(camX, camY, camZ, // Posicion de la camara
              pickObjX, pickObjY, pickObjZ, // Mirando al punto
              0.0, 1.0, 0.0); // Up vector
}

void GL_dibujarCuerpos(void) {
    int i;

    for (i = 0; i < N; i++) {
        glPushMatrix();
        glTranslatef(cPositionX[i], cPositionY[i], cPositionZ[i]);

        //reemplazar por los valores random de los colores//

        glColor3f((double) rand() / (RAND_MAX + 1.0), (double) rand() / (RAND_MAX + 1.0), (double) rand() / (RAND_MAX + 1.0));
        glutSolidSphere(0.02, 20, 20);

        glPopMatrix();
    }

    //ACA!!! se Llama a la funcion que calcula las fuerzas nuevamente
    //gravitacion GPU//
    gravitacionGPU(N, delta_tiempo);
    //TRAERME LOS DATOS DE LA GPU//

    hipMemcpy(cPositionX, gpu_cPositionX, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cPositionY, gpu_cPositionY, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cPositionZ, gpu_cPositionZ, N * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(cVelocityX, gpu_cVelocityX, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cVelocityY, gpu_cVelocityY, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cVelocityZ, gpu_cVelocityZ, N * sizeof(float), hipMemcpyDeviceToHost);

    //gravitacionCPU(N,delta_tiempo);
}

void GL_dibujar(void) {
    // Borra el color y los buffers de profundidad
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // Reiniciar la matriz de transformaciones
    glLoadIdentity();

    //ubica la camara
    GL_camara();

    //Dibuja los ejes de coordenadas (si estan habilitados)
    if (ejes) {
        glBegin(GL_LINES);
        glColor3f(1.0, 0.0, 0.0);
        glVertex3d(0, 0, 0);
        glVertex3d(5.0, 0.0, 0.0);

        glColor3f(0.0, 1.0, 0.0);
        glVertex3d(0, 0, 0);
        glVertex3d(0.0, 5.0, 0.0);

        glColor3f(0.0, 0.0, 1.0);
        glVertex3d(0, 0, 0);
        glVertex3d(0, 0, 5.0);

        glEnd();
    }

    // Dibuja
    glPushMatrix();
    GL_dibujarCuerpos();
    glPopMatrix();

    glutSwapBuffers();
}

void GL_cambioDeDimensionDeVentana(int w, int h) {

    // Evita que se divida por cero cuando la ventana es muy chica
    if (h == 0) h = 1;
    float ratio = w * 1.0 / h;

    // Usa la matriz de proyecion
    glMatrixMode(GL_PROJECTION);

    // Reset matriz
    glLoadIdentity();

    // Configura el viewport para la ventana completa
    glViewport(0, 0, w, h);

    // Configura la perspectiva correcta
    gluPerspective(45.0f, ratio, 0.1f, 100.0f);

    // Modelview
    glMatrixMode(GL_MODELVIEW);
}

//Funcion de inicializacion
void GL_inicio(void) {
    glClearColor(0.0, 0.0, 0.0, 0.0);
    glOrtho(-10, 10, -10, 10, -10, 10);
}

void GL_teclado(unsigned char key, int x, int y) {
    double denominador = 50.0;
    double grados = PI / denominador;
    switch (key) {
    case 'a':
        if (alfa + grados >= 2 * PI)
            alfa = (alfa + grados) - 2 * PI;
        else
            alfa += grados;
        break;
    case '+':
        distancia--;
        break;
    case '-':
        distancia++;
        break;
    case 'e':
        if (ejes == 1) {
            ejes = 0;
        } else {
            ejes = 1;
        }
        break;
    case KEY_ESC:
        finalizar();
        exit(0); // Sale de la aplicacion si se presiona 'Esc'
    }
    glutPostRedisplay();
}

void GL_teclasEspeciales(int key, int x, int y) {
    double denominador = 50.0;
    double grados = PI / denominador;

    switch (key) {
    case GLUT_KEY_RIGHT:
        if (camAngleX - grados < 0)
            camAngleX = (camAngleX - grados) + 2 * PI;
        else
            camAngleX -= grados;
        break;
    case GLUT_KEY_LEFT:
        if (camAngleX + grados >= 2 * PI)
            camAngleX = (camAngleX + grados) - 2 * PI;
        else
            camAngleX += grados;
        break;
    case GLUT_KEY_UP:
        if (camAngleY - grados <= -PI / 2)
            camAngleY = -PI / 2 + 0.001;
        else
            camAngleY -= grados;
        break;
    case GLUT_KEY_DOWN:
        if (camAngleY + grados >= PI / 2)
            camAngleY = PI / 2 - 0.001;
        else
            camAngleY += grados;
        break;
    }

    glutPostRedisplay();
}

void GL_OnMouseDown(int button, int state, int x, int y) {
    rotate = false;
    if (button == GLUT_LEFT_BUTTON) {
        oldX = x;
        oldY = y;
        rotate = true;
    }

}

void GL_OnMouseMove(int x, int y) {

    if (rotate) {
        camAngleX -= (x - oldX) * 0.01f;
        camAngleY += (y - oldY) * 0.01f;
    }

    oldX = x;
    oldY = y;
    glutPostRedisplay();
}

void procesoOpenGL(int argc, char * argv[]) {
    //Inicializa la libreria glut
    glutInit( & argc, argv);
    //Se va a usar doble buffer, paleta RGB
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    //Define la ventana de visualizacion
    glutInitWindowSize(ANCHO, ALTO);

    //Posicionar la ventana
    glutInitWindowPosition(0, 0);
    //Se crea la ventana cuyo nombre en la barra de titulo es lo que viene en argv[0]
    glutCreateWindow(argv[0]);

    //Funcion personalizada que inicializa parametros
    GL_inicio();

    //Define cual es la funcion de control de renderizado
    // Se llama cada vez que se quiere dibujar nuevamente en la pantalla (cada vez que se produce el evento render)
    //GL DIBUJAR LLAMA A NUESTRO CÓDIGO//
    glutDisplayFunc(GL_dibujar);
    glutReshapeFunc(GL_cambioDeDimensionDeVentana);
    glutIdleFunc(GL_dibujar);

    //Define cuales son las funciones que atenderan los eventos del teclado
    glutKeyboardFunc(GL_teclado);
    glutSpecialFunc(GL_teclasEspeciales);

    //Define cuales son las funciones que atenderan los eventos del mouse
    glutMouseFunc(GL_OnMouseDown);
    glutMotionFunc(GL_OnMouseMove);

    //El programa espera aca
    glutMainLoop();
}

int main(int argc, char * argv[]) {

    N = atoi(argv[1]);
    delta_tiempo = atof(argv[2]);
    pasos = atoi(argv[3]);

    //CPU VARIABLES
    cPositionX = (float * ) malloc(N * sizeof(float));
    cPositionY = (float * ) malloc(N * sizeof(float));
    cPositionZ = (float * ) malloc(N * sizeof(float));

    cVelocityX = (float * ) malloc(N * sizeof(float));
    cVelocityY = (float * ) malloc(N * sizeof(float));
    cVelocityZ = (float * ) malloc(N * sizeof(float));

    masas = (float * ) malloc(N * sizeof(float));

    fuerza_totalX = (float * ) malloc(sizeof(float) * N);
    fuerza_totalY = (float * ) malloc(sizeof(float) * N);
    fuerza_totalZ = (float * ) malloc(sizeof(float) * N);

    //GPU VARIABLES
    gpu_cPositionX = (float * ) malloc(N * sizeof(float));
    gpu_cPositionY = (float * ) malloc(N * sizeof(float));
    gpu_cPositionZ = (float * ) malloc(N * sizeof(float));

    gpu_cVelocityX = (float * ) malloc(N * sizeof(float));
    gpu_cVelocityY = (float * ) malloc(N * sizeof(float));
    gpu_cVelocityZ = (float * ) malloc(N * sizeof(float));

    gpu_masas = (float * ) malloc(N * sizeof(float));

    gpu_fuerza_totalX = (float * ) malloc(sizeof(float) * N);
    gpu_fuerza_totalY = (float * ) malloc(sizeof(float) * N);
    gpu_fuerza_totalZ = (float * ) malloc(sizeof(float) * N);

    hipMalloc( & gpu_cPositionX, N * sizeof(float));
    hipMalloc( & gpu_cPositionY, N * sizeof(float));
    hipMalloc( & gpu_cPositionZ, N * sizeof(float));

    hipMalloc( & gpu_cVelocityX, N * sizeof(float));
    hipMalloc( & gpu_cVelocityY, N * sizeof(float));
    hipMalloc( & gpu_cVelocityZ, N * sizeof(float));

    hipMalloc( & gpu_masas, N * sizeof(float));

    hipMalloc( & gpu_fuerza_totalX, N * sizeof(float));
    hipMalloc( & gpu_fuerza_totalY, N * sizeof(float));
    hipMalloc( & gpu_fuerza_totalZ, N * sizeof(float));

    inicializarCuerpos(N);

    //aca pasamos los datos a la GPU por primera vez//
    hipMemcpy(gpu_cPositionX, cPositionX, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cPositionY, cPositionY, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cPositionZ, cPositionZ, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(gpu_cVelocityX, cVelocityX, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cVelocityY, cVelocityY, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cVelocityZ, cVelocityZ, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(gpu_masas, masas, N * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(gpu_fuerza_totalX, fuerza_totalX, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_fuerza_totalY, fuerza_totalY, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_fuerza_totalZ, fuerza_totalZ, N * sizeof(float), hipMemcpyHostToDevice);

    //ADENTRO DE ESTO SE VA A LLAMAR AL CALCULO//
    procesoOpenGL(argc, argv);

    return (0);

}