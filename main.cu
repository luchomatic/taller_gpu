#include "hip/hip_runtime.h"
#include <GL/glut.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>
#include <math.h>

//
// Constantes para OpenGL
//
#define KEY_ESC 27
#define ANCHO 1920
#define ALTO 1080
#define false 0
#define true 1

//
// Constantes para Algoritmo de gravitacion
//
#define PI (3.141592653589793)
#define G 6.673e-11


// ===============
// ===== CPU =====
// ===============

//
// Estructuras y variables para Algoritmo de gravitacion
//



float toroide_alfa;
float toroide_theta;
float toroide_incremento;
float toroide_lado;
float toroide_r;
float toroide_R;


int delta_tiempo = 1.0f; //Intervalo de tiempo, longitud de un paso
int pasos;
int N;

//variables nuestras re locas//
float *masas;
float *cPositionX;
float *cPositionY;
float *cPositionZ;

float *cVelocityX;
float *cVelocityY;
float *cVelocityZ;

float *fuerza_totalX;
float *fuerza_totalY; 
float *fuerza_totalZ;


double cColorR = (double )rand()/(RAND_MAX+1.0);
double cColorG = (double )rand()/(RAND_MAX+1.0);
double cColorB = (double )rand()/(RAND_MAX+1.0);

///terminan las variables nuestras re locas dodi.//

//
// Funciones para Algoritmo de gravitacion
//

void calcularFuerzas(int N, int dt){
int cuerpo1, cuerpo2;
float dif_X, dif_Y, dif_Z;
float distancia;
float F;

	for(cuerpo1 = 0; cuerpo1<N-1 ; cuerpo1++){
		for(cuerpo2 = cuerpo1 + 1; cuerpo2<N ; cuerpo2++){
			if ( (cPositionX[cuerpo1] == cPositionX[cuerpo2]) && (cPositionY[cuerpo1]== cPositionY[cuerpo2]) && (cPositionZ[cuerpo1] == cPositionZ[cuerpo2]))
                continue;

	            	dif_X = cPositionX[cuerpo2] - cPositionX[cuerpo1];
			dif_Y = cPositionY[cuerpo2] - cPositionY[cuerpo1];
			dif_Z = cPositionZ[cuerpo2] - cPositionZ[cuerpo1];
                
			distancia = sqrt(dif_X*dif_X + dif_Y*dif_Y + dif_Z*dif_Z);

	                F = (G*masas[cuerpo1]*masas[cuerpo2])/(distancia*distancia);

	                dif_X *= F;
			dif_Y *= F;
			dif_Z *= F;

	                fuerza_totalX[cuerpo1] += dif_X;
	                fuerza_totalY[cuerpo1] += dif_Y;
	                fuerza_totalZ[cuerpo1] += dif_Z;

	                fuerza_totalX[cuerpo2] -= dif_X;
	                fuerza_totalY[cuerpo2] -= dif_Y;
	                fuerza_totalZ[cuerpo2] -= dif_Z;
		}
	}
}

void moverCuerpos(int N, int dt){
 int cuerpo;
	for(cuerpo = 0; cuerpo<N ; cuerpo++){

        	fuerza_totalX[cuerpo] *= 1/masas[cuerpo];
        	fuerza_totalY[cuerpo] *= 1/masas[cuerpo];
        	

        	cVelocityX[cuerpo] += fuerza_totalX[cuerpo]*dt;
        	cVelocityY[cuerpo] += fuerza_totalY[cuerpo]*dt;
        	

        	cPositionX[cuerpo] += cVelocityX[cuerpo] *dt;
        	cPositionY[cuerpo] += cVelocityY[cuerpo] *dt;
        	

        	fuerza_totalX[cuerpo] = 0.0;
		fuerza_totalY[cuerpo] = 0.0;
		fuerza_totalZ[cuerpo] = 0.0;

    	}
}

void gravitacionCPU(int N, int dt){
	//reescribir estas dos funciones en gpu//
	calcularFuerzas(N,dt);
	moverCuerpos(N,dt);
}

void inicializarEstrella(int i,double n){
	
    //todos van a tener la misma masa//
    masas[i] = 0.001*8;

        if ((toroide_alfa + toroide_incremento) >=2*M_PI){
            toroide_alfa = 0;
            toroide_theta += toroide_incremento;
        }else{
            toroide_alfa+=toroide_incremento;
        }

	cPositionX[i] = (toroide_R + toroide_r*cos(toroide_alfa))*cos(toroide_theta); 
	cPositionY[i] = (toroide_R + toroide_r*cos(toroide_alfa))*sin(toroide_theta);
 	cPositionZ[i] = toroide_r*sin(toroide_alfa);

	cVelocityX[i] = 0.0;
	cVelocityY[i] = 0.0;
	cVelocityZ[i] = 0.0;
}



void inicializarCuerpos(int N){
 int cuerpo;
 double n = N;

	

	toroide_alfa = 0.0;
	toroide_theta = 0.0;
	toroide_lado = sqrt(N);
	toroide_incremento = 2*M_PI / toroide_lado;
	toroide_r = 1.0;
	toroide_R = 2*toroide_r;
	
	srand(time(NULL));

	for(cuerpo = 0; cuerpo < N; cuerpo++){

        	fuerza_totalX[cuerpo] = 0.0;
		fuerza_totalY[cuerpo] = 0.0;
		fuerza_totalZ[cuerpo] = 0.0;

		inicializarEstrella(cuerpo,n);
		
	}
		masas[0] = 2.0e2;
	        cPositionX[0] = 0.0;
		cPositionY[0] = 0.0;
		cPositionZ[0] = 0.0;
		cVelocityX[0] = -0.000001;
		cVelocityY[0] = -0.000001;
		cVelocityZ[0] = 0.0;

		masas[1] = 1.0e1;
	        cPositionX[1] = -1.0;
		cPositionY[1] = 0.0;
		cPositionZ[1] = 0.0;
		cVelocityX[1] = 0.0;
		cVelocityY[1] = 0.0001;
		cVelocityZ[1] = 0.0;

}

void finalizar(void){
	free(masas);
	free(cPositionX);
	free(cPositionY);
	free(cPositionZ);
	free(cVelocityX);
	free(cVelocityY);
	free(cVelocityZ);
	free(fuerza_totalX);
	free(fuerza_totalY);
	free(fuerza_totalZ);
}

// ===============
// ===== GPU =====
// ===============

__global__ void kernelGravitacion(void){
 printf("Hello\n");
}

void gravitacionGPU( int N, int dt){
 	
	kernelGravitacion<<<1,256>>>();	
}

// ==================
// ===== OpenGL =====
// ==================

//
// Variables OpenGL
//
double alfa=0.0;

// Para angulo de rotacion y direccion de la camara
float angle=0.0;
float camAngleX=0;
float camAngleY=0;
float distancia=10;
int ejes = 1;

// Vector actual que representa la direccion de la camara
float lx=0.0f,lz=-1.0f;
// posicion XZ de la camara
float x=0.0f,z=5.0f;

int oldX=0, oldY=0;
int rotate = false;

//
// Funciones OpenGL
//

//Funcion que se llama cada vez que se quiere dibujar nuevamente en la pantalla
//Se llama cada vez que se produce el evento render
void GL_camara(){
 float camX,camY,camZ;

	//Camara mirando al origen (pickObjX,pickObjY,pickObjZ) = (0,0,0)
	float pickObjX = 0.0;
	float pickObjY = 0.0;
	float pickObjZ = 0.0;

	camX = distancia * sin(camAngleX);
	camY = distancia * sin(camAngleY);
	camZ = distancia * cos(camAngleY)*cos(camAngleX);

	//Ubicar la camara
	gluLookAt(camX,camY,camZ,   // Posicion de la camara
          pickObjX,pickObjY,pickObjZ,    // Mirando al punto
          0.0, 1.0, 0.0);   // Up vector
}

void GL_dibujarCuerpos(void){
int i;

	 for(i=0;i<N;i++){
	  glPushMatrix();
	  glTranslatef(cPositionX[i],cPositionY[i],cPositionZ[i]);
	  

	  //reemplazar por los valores random de los colores//
	
	  glColor3f((double )rand()/(RAND_MAX+1.0),(double )rand()/(RAND_MAX+1.0),(double )rand()/(RAND_MAX+1.0));	        
          glutSolidSphere(0.02, 20, 20);
        	
          glPopMatrix();
	}

	//ACA!!! se Llama a la funcion que calcula las fuerzas nuevamente
	//gravitacion GPU//
	//gravitacionGPU(cuerpos,N,delta_tiempo);
	//TRAERME LOS DATOS DE LA GPU//
	gravitacionCPU(N,delta_tiempo);
}

void GL_dibujar(void) {
	// Borra el color y los buffers de profundidad
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// Reiniciar la matriz de transformaciones
	glLoadIdentity();

	//ubica la camara	
	GL_camara();

	    //Dibuja los ejes de coordenadas (si estan habilitados)
	    if (ejes){
		    glBegin(GL_LINES);
		    glColor3f(1.0,0.0,0.0);
		    glVertex3d(0,0,0);
	 	    glVertex3d(5.0,0.0,0.0);
	
        	    glColor3f(0.0,1.0,0.0);
	            glVertex3d(0,0,0);
	            glVertex3d(0.0,5.0,0.0);

	            glColor3f(0.0,0.0,1.0);
	            glVertex3d(0,0,0);
	            glVertex3d(0,0,5.0);

		    glEnd();
	    }

	// Dibuja
	glPushMatrix();
	GL_dibujarCuerpos();
	glPopMatrix();

	glutSwapBuffers();
}

void GL_cambioDeDimensionDeVentana(int w, int h) {

	// Evita que se divida por cero cuando la ventana es muy chica
	if (h == 0) h = 1;
	float ratio = w * 1.0 / h;

	// Usa la matriz de proyecion
	glMatrixMode(GL_PROJECTION);

	// Reset matriz
	glLoadIdentity();

	// Configura el viewport para la ventana completa
	glViewport(0, 0, w, h);

	// Configura la perspectiva correcta
	gluPerspective(45.0f, ratio, 0.1f, 100.0f);

	// Modelview
	glMatrixMode(GL_MODELVIEW);
}

//Funcion de inicializacion
void GL_inicio(void){
    glClearColor(0.0,0.0,0.0,0.0);
    glOrtho(-10,10,-10,10,-10,10);
}

void GL_teclado(unsigned char key, int x, int y) {
double denominador=50.0;
double grados = PI/denominador;
	switch (key) {
        case 'a':
            if (alfa + grados >= 2*PI)
                alfa = (alfa + grados) - 2*PI;
            else
                alfa += grados;
            break;
        case '+':
            distancia--;
            break;
        case '-':
            distancia++;
            break;
        case 'e':
            if (ejes==1) {ejes=0;} else{ejes=1;}
            break;
		case KEY_ESC:
			finalizar();
			exit(0); // Sale de la aplicacion si se presiona 'Esc'
	}
	glutPostRedisplay();
}

void GL_teclasEspeciales(int key, int x, int y){
 double denominador=50.0;
 double grados = PI/denominador;

	switch (key) {
	    case GLUT_KEY_RIGHT :
	        if (camAngleX - grados < 0)
	            camAngleX = (camAngleX - grados) + 2*PI;
        	else
        	    camAngleX -= grados;
        	break;
	    case GLUT_KEY_LEFT :
        	if (camAngleX + grados >= 2*PI)
        	    camAngleX = (camAngleX + grados) - 2*PI;
	        else
        	    camAngleX += grados;
        	break;
	    case GLUT_KEY_UP :
        	if (camAngleY - grados <= -PI/2)
        	    camAngleY = -PI/2 + 0.001;
        	else
        	    camAngleY -= grados;
        	break;
	    case GLUT_KEY_DOWN :
        	if (camAngleY + grados >= PI/2)
        	    camAngleY = PI/2 - 0.001;
        	 else
        	    camAngleY += grados;
        	break;
	}

  	glutPostRedisplay();
}

void GL_OnMouseDown(int button, int state, int x, int y) {
   rotate=false;
   if(button == GLUT_LEFT_BUTTON) {
      oldX = x;
      oldY = y;
      rotate = true;
   }

}

void GL_OnMouseMove(int x, int y) {

   if(rotate) {
      camAngleX -= (x-oldX)*0.01f;
      camAngleY   += (y-oldY)*0.01f;
   }

   oldX = x;
   oldY = y;
   glutPostRedisplay();
}

void procesoOpenGL(int argc, char * argv[]){
   //Inicializa la libreria glut
    glutInit(&argc, argv);
    //Se va a usar doble buffer, paleta RGB
    glutInitDisplayMode (GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
    //Define la ventana de visualizacion
    glutInitWindowSize (ANCHO, ALTO);

    //Posicionar la ventana
    glutInitWindowPosition(0,0);
    //Se crea la ventana cuyo nombre en la barra de titulo es lo que viene en argv[0]
    glutCreateWindow (argv[0]);

    //Funcion personalizada que inicializa parametros
    GL_inicio();

    //Define cual es la funcion de control de renderizado
    // Se llama cada vez que se quiere dibujar nuevamente en la pantalla (cada vez que se produce el evento render)
    //GL DIBUJAR LLAMA A NUESTRA PORQUERIA//
    glutDisplayFunc (GL_dibujar);
    glutReshapeFunc(GL_cambioDeDimensionDeVentana);
    glutIdleFunc(GL_dibujar);

    //Define cuales son las funciones que atenderan los eventos del teclado
    glutKeyboardFunc (GL_teclado);
    glutSpecialFunc(GL_teclasEspeciales);

    //Define cuales son las funciones que atenderan los eventos del mouse
    glutMouseFunc(GL_OnMouseDown);
    glutMotionFunc(GL_OnMouseMove);

    //El programa espera aca
    glutMainLoop();
}


int main(int argc, char * argv[]) {

	N = atoi(argv[1]);
	delta_tiempo = atof(argv[2]);
	pasos = atoi(argv[3]);
	

	cPositionX = (float*) malloc (N*sizeof(float));  
	cPositionY = (float*) malloc (N*sizeof(float));
	cPositionZ = (float*) malloc (N*sizeof(float));
	
	cVelocityX = (float*) malloc (N*sizeof(float));
	cVelocityY = (float*) malloc (N*sizeof(float));
	cVelocityZ = (float*) malloc (N*sizeof(float));
	
	masas = (float*) malloc (N*sizeof(float));

	
	fuerza_totalX = (float*)malloc(sizeof(float)*N);
	fuerza_totalY = (float*)malloc(sizeof(float)*N);
	fuerza_totalZ = (float*)malloc(sizeof(float)*N);

	inicializarCuerpos(N);
	

	//aca pasamos los datos a la GPU por primera vez//
	//ADENTRO DE ESTO SE VA A LLAMAR AL CALCULO//
	procesoOpenGL(argc,argv);

    return(0);

}
